
#include <hip/hip_runtime.h>

__global__ void mulf32(float* x1, float* x2, float* y) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    y[index] = x1[index] * x2[index];
}
