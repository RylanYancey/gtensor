
#include <hip/hip_runtime.h>

__global__ void mulf64(double* x1, double* x2, double* y) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    y[index] = x1[index] * x2[index];
}