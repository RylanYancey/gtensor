

#include <hip/hip_runtime.h>
#include<hip/hip_fp16.h>

__global__ void mulf16(__half* x1, __half* x2, __half* y) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    y[index] = __hmul(x1[index], x2[index]);
}