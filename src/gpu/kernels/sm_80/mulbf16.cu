

#include <hip/hip_runtime.h>
#include<hip/hip_bf16.h>

__global__ void mulbf16(__hip_bfloat16* x1, __hip_bfloat16* x2, __hip_bfloat16* y) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    y[index] = __hmul(x1[index], x2[index]);
}